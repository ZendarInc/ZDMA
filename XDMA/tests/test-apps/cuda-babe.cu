#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <errno.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/ioctl.h>
#include <sys/stat.h>
#include <sys/types.h>

#define SURFACE_W  256
#define SURFACE_H  256
#define SURFACE_SIZE  (SURFACE_W * SURFACE_H)

#define OFFSET(x, y)  (((y) * SURFACE_W) + x)
#define DATA(x, y)  (((y & 0xffff) << 16) | ((x) & 0xffff))

extern "C" __global__ void reorder_bytes(uint32_t* gpu_data)
{
  unsigned int pos_x = (blockIdx.x * blockDim.x) + threadIdx.x;
  unsigned int pos_y = (blockIdx.y * blockDim.y) + threadIdx.y;

  const uint32_t word = gpu_data[OFFSET(pos_x, pos_y)];
  const uint8_t b0 = word & 0xFF;
  const uint8_t b1 = (word >> 8) & 0xFF;
  const uint8_t b2 = (word >> 16) & 0xFF;
  const uint8_t b3 = (word >> 24) & 0xFF;

  gpu_data[OFFSET(pos_x, pos_y)] = (b0 << 24) | (b1 << 16) | (b2 << 8) | b3;
}

void HexDump(const uint8_t* bytes, size_t size)
{
  if (!size) return;

  const size_t bytes_per_line = 16;
  const size_t total_lines = ((size - 1) / bytes_per_line) + 1;

  for (size_t line = 0; line < total_lines; ++line) {
    const unsigned int offset = line * bytes_per_line;
    // Show the offset each line:
    printf("%08x", offset);

    // Hex bytes
    for (size_t i = offset; i < offset + bytes_per_line; ++i) {
      // Add a bit of space for visual clarity
      if (i % (bytes_per_line / 2) == 0)
        printf(" ");
      if (i < size)
        printf(" %02x", bytes[i]);
      else
        printf("   ");
    }

    // printable characters
    printf("  ");
    for (size_t i = offset; i < offset + bytes_per_line && i < size; ++i) {
      char c = bytes[i];
      const char first_printable = ' ';
      const char last_printable = '\x7e';
      if (c < first_printable || c > last_printable)
        printf(".");
      else
        printf("%c", c);
    }
    printf("\n");
    fflush(stdout);
  }
}

uint32_t init_data[SURFACE_SIZE * sizeof(uint32_t)];

int main(int argc, char **argv)
{
  hipError_t ce;
  hipError_t cr;
  uint32_t* src_d;
  int c2h_fd, h2c_fd, ret;
  unsigned int flag = 1;
  /*
  struct picoevb_rdma_pin_cuda pin_params_src;
  struct picoevb_rdma_h2c2h_dma dma_params;
  struct picoevb_rdma_unpin_cuda unpin_params_src;
  */

  if (argc != 1) {
    fprintf(stderr, "usage: cuda-babe\n");
    return 1;
  }

  //c2h_fd = open("/dev/picoevb", O_RDWR);
  c2h_fd = open("/dev/xdma0_c2h_0", O_RDONLY);
  if (c2h_fd < 0) {
    perror("open() failed");
    return 1;
  }

  h2c_fd = open("/dev/xdma0_h2c_0", O_WRONLY);
  if (h2c_fd < 0) {
    perror("open() failed");
    return 1;
  }

  // *** INIT DATA to FPGA ***
  for (size_t i = 0; i < SURFACE_SIZE; ++i) {
    init_data[i] = i;
  }

  ret = lseek(h2c_fd, 0x80000000, SEEK_SET);
  if (ret == -1) {
    fprintf(stderr, "lseek(DMA) failed: %d\n", ret);
    perror("lseek() failed");
    return 1;
  }
  ret = write(h2c_fd, init_data, SURFACE_SIZE * sizeof(uint32_t));
  if (ret == -1) {
    fprintf(stderr, "write(DMA) failed: %d\n", ret);
    perror("write() failed");
    return 1;
  }
  close(h2c_fd);

  ce = hipMallocManaged(&src_d, SURFACE_SIZE * sizeof(*src_d));

  if (ce != hipSuccess) {
    fprintf(stderr, "Allocation of src_d failed: %s\n",
      hipGetErrorString(ce));
    return 1;
  }

  cr = hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS,
    (hipDeviceptr_t)src_d);
  if (cr != hipSuccess) {
    fprintf(stderr, "cuPointerSetAttribute(src_d) failed: %d\n", cr);
    return 1;
  }

  ret = lseek(c2h_fd, 0x80000000, SEEK_SET);
  if (ret == -1) {
    fprintf(stderr, "lseek(DMA) failed: %d\n", ret);
    perror("lseek() failed");
    return 1;
  }

  ret = read(c2h_fd, src_d, SURFACE_SIZE * sizeof(*src_d));
  if (ret == -1) {
    fprintf(stderr, "read(DMA) failed: %d\n", ret);
    perror("read() failed");
    return 1;
  }

  dim3 dimGrid(SURFACE_W / 16, SURFACE_H / 16);
  dim3 dimBlock(16, 16);
  reorder_bytes<<<dimGrid, dimBlock>>>(src_d);

  ce = hipDeviceSynchronize();
  if (ce != hipSuccess) {
    fprintf(stderr, "cudaDeviceSynchronize() failed: %d\n", ce);
    return 1;
  }

  // If this works, it's because of some weird zero-copy logic.
  HexDump((uint8_t*)src_d, SURFACE_SIZE * sizeof(*src_d));

  ce = hipFree(src_d);

  if (ce != hipSuccess) {
    fprintf(stderr, "Free of src_d failed: %d\n", ce);
    return 1;
  }

  ret = close(c2h_fd);
  if (ret < 0) {
    perror("close() failed");
    return 1;
  }
}

